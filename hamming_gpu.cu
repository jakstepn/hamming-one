#include "hip/hip_runtime.h"
#include <iostream>
#include "output.h"
#include "word_reader.h"
#include "hamming_gpu.cuh"
#include <algorithm>
#include "hip/hip_runtime.h"
#include ""

// unsigned long long 64 bits
#define LOOKUP_SIZE 256
#define MAX_PAIRS 10

#define check_err(ans)                                                                              \
        if (ans != hipSuccess)																		\
        {																							\
            fprintf(stderr, "Assert: %s %s %d\n", hipGetErrorString(ans), __FILE__, __LINE__);		\
            exit(ans);																			\
        }																							\

__device__ unsigned long long distance_between_fragments(int fragment1, int fragment2, int* bit_count)
{
	// Xor of two parts of a vector
	int to_check = fragment1 ^ fragment2;
	return static_cast<unsigned long long>(bit_count[to_check & 0xff] + bit_count[(to_check >> 8) & 0xff] +
		bit_count[(to_check >> 16) & 0xff] + bit_count[(to_check >> 24) & 0xff]);
}

__device__ int word_elem_index(int iteration, int lines_amount, int id)
{
	// Fragment index inside a vector array
	return iteration * lines_amount + id;
}

__global__ void analyze_lines(int* vectors, int vector_len, int lines_amount,
	int* pairs_vec, unsigned long long* amount_of_pairs, bool find_pairs,
	int* bit_count)
{
	// Global thread id
	int gid = blockDim.x * blockIdx.x + threadIdx.x;
	unsigned long long distance = 0;
	int elem_index, next_elem_index, pair_count = 0;

	if (gid < lines_amount)
	{
		// For every other word
		for (int j = gid + 1; j < lines_amount; j++)
		{
			// For every int
			for (int i = 0; i < vector_len; i++)
			{
				elem_index = word_elem_index(i, lines_amount, gid);
				next_elem_index = word_elem_index(i, lines_amount, j);
				distance += distance_between_fragments(vectors[elem_index], vectors[next_elem_index], bit_count);
			}
			if (distance == 1)
			{
				pair_count++;
				if (find_pairs && pair_count <= MAX_PAIRS)
				{
					pairs_vec[gid * MAX_PAIRS + pair_count - 1] = j;
				}
			}
			distance = 0;
		}
		atomicAdd(amount_of_pairs, pair_count);
	}

	__syncthreads();
}

void main_gpu(bool find_pairs, FileReader* reader, int* lookup_table)
{
	int number_of_vectors = reader->amount_of_vectors,
		vector_length = reader->vector_int_length;

	unsigned long long* amount_of_pairs = new unsigned long long(0);

	// Array for storing pairs
	int* res = new int[number_of_vectors * MAX_PAIRS];
	try 
	{
		std::fill(res, res + number_of_vectors * MAX_PAIRS, -1);
	}
	catch (const std::bad_alloc& e) {
		fprintf(stderr, "%s", e.what());
	}

	int* d_vectors;
	unsigned long long* d_amount_of_pairs;
	int* d_lookup_table;
	int* d_res;

	int num_threads = 1024, num_blocks;
	num_blocks = (number_of_vectors + num_threads - 1) / num_threads;

	// Timers
	float time;
	hipEvent_t start, stop;

	check_err(hipEventCreate(&start));
	check_err(hipEventCreate(&stop));

	// Device memory allocation
	check_err(hipMalloc(&d_vectors, sizeof(int) * vector_length * number_of_vectors));
	check_err(hipMalloc(&d_res, sizeof(int) * number_of_vectors * MAX_PAIRS));
	check_err(hipMalloc(&d_amount_of_pairs, sizeof(unsigned long long)));
	check_err(hipMalloc(&d_lookup_table, sizeof(int) * LOOKUP_SIZE));

	check_err(hipMemcpy(d_res, res, sizeof(int) * number_of_vectors * MAX_PAIRS, hipMemcpyHostToDevice));
	check_err(hipMemcpy(d_vectors, reader->vectors, sizeof(int) * number_of_vectors * vector_length, hipMemcpyHostToDevice));
	check_err(hipMemcpy(d_amount_of_pairs, amount_of_pairs, sizeof(unsigned long long), hipMemcpyHostToDevice));
	check_err(hipMemcpy(d_lookup_table, lookup_table, sizeof(int) * LOOKUP_SIZE, hipMemcpyHostToDevice));

	check_err(hipEventRecord(start, 0));

	analyze_lines<<<num_blocks, num_threads>>>(d_vectors, vector_length, number_of_vectors,
		d_res, d_amount_of_pairs, find_pairs, d_lookup_table);
	check_err(hipDeviceSynchronize());

	check_err(hipEventRecord(stop, 0));
	check_err(hipEventSynchronize(stop));
	check_err(hipEventElapsedTime(&time, start, stop));

	check_err(hipEventDestroy(start));
	check_err(hipEventDestroy(stop));

	check_err(hipMemcpy(amount_of_pairs, d_amount_of_pairs, sizeof(unsigned long long), hipMemcpyDeviceToHost));

	std::cout << "Pairs: " << *amount_of_pairs << std::endl;
	std::cout << "Calculation time (ms): " << time << std::endl;

	check_err(hipMemcpy(res, d_res, sizeof(int) * number_of_vectors * MAX_PAIRS, hipMemcpyDeviceToHost));

	if (find_pairs)
		print_console_vector(reader->vectors, res, vector_length, number_of_vectors, MAX_PAIRS);

	check_err(hipFree(d_vectors));
	check_err(hipFree(d_res));
	check_err(hipFree(d_amount_of_pairs));
	check_err(hipFree(d_lookup_table));

	delete[] res;
	delete amount_of_pairs;
}